
#include "hip/hip_runtime.h"
#include ""
#include <random>
#include <iostream>
#include "matmul.cuh"
#include "queue.h"
#include <chrono>

const int sizeX = 100;
const int sizeY = 100;

template <typename T>
void showResultsCuda(T* a, T* b, T*c, int sizeX, int sizeY, bool showMatrices) {
	auto start = chrono::high_resolution_clock::now();
	hipError_t cudaStatus = cudaMatmul(a, b, c, sizeX, sizeY);
	ReportError(cudaStatus != hipSuccess, "Error of running cudaMatmul!");
	auto end = chrono::high_resolution_clock::now();
	chrono::duration<double> duration = end - start;
	if (showMatrices) {
		cout << "matrix a:" << "\n\n";
		showMatrix(a, sizeX, sizeY);
		cout << "matrix b:" << "\n\n";
		showMatrix(b, sizeX, sizeY);
		cout << "\n\n" << "cuda time of calculating: " << duration.count() << "\n";
		cout << "matrix c_cuda:" << "\n\n";
		showMatrix(c, sizeX, sizeY);
	}
	else {
		cout << "\n\n" << "cuda time of calculating: " << duration.count() << "\n";
	}
}

int main() {
	double mean = 0;
	double sd = 1;
	double* a = new double[sizeX * sizeY];
	double* b = new double[sizeX * sizeY];
	double* c_cuda = new double[sizeX * sizeY];
	double* c_queue = new double[sizeX * sizeY];
	for (int i = 0; i < sizeX; i++) {
		for (int j = 0; j < sizeY; j++) {
			a[i * sizeY + j] = random(mean, sd);
			b[i * sizeY + j] = random(mean, sd);
			c_cuda[i * sizeY + j] = 0;
			c_queue[i * sizeY + j] = 0;
		}
	}
	// code for parallel calculating matrix multiplication ... 
	showResultsCuda(a, b, c_cuda, sizeX, sizeY, false);
	queueMatmul(a, b, c_queue, sizeX, sizeY, false);
	if (checker(c_cuda, c_queue, sizeX, sizeY)) {
		cout << "\n" << "c_cuda equal c_queue" << "\n";
	}
	else {
		cout << "\n" << "c_cuda not equal c_queue" << "\n";
	}
	return 0;
}